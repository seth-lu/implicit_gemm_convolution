#include "hip/hip_runtime.h"
#include<cstdlib>
#include<cstdio>
#include "hipDNN.h"
#include "implicit_gemm_kernel.h"

#define ErrChk(code) { Assert((code), __FILE__, __LINE__); }
inline void Assert(hipError_t  code, const char *file, int line){
	if(code!=hipSuccess) {
		printf("CUDA Runtime Error: %s:%d:'%s'\n", file, line, hipGetErrorString(code));
		exit(EXIT_FAILURE);
	}
}
inline void Assert(hipdnnStatus_t code, const char *file, int line){
    if (code!=HIPDNN_STATUS_SUCCESS){
		printf("cuDNN API Error: %s:%d:'%s'\n", file, line, hipdnnGetErrorString(code));
        exit(EXIT_FAILURE);
    }
}

#define KernelErrChk(){\
		hipError_t errSync  = hipGetLastError();\
		hipError_t errAsync = hipDeviceSynchronize();\
		if (errSync != hipSuccess) {\
			  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));\
			  exit(EXIT_FAILURE);\
		}\
		if (errAsync != hipSuccess){\
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));\
			exit(EXIT_FAILURE);\
		}\
}


int main(int argc, char *argv[]){
	
	//convolution parameters
	int N = 32; //batch size
	int C = 1024; //channel
	int H = 24;
	int W = 24;
	
	int K = 16; //number of filter
	int R = 1;
	int S = 1;
	int U = 1; //stride_h
	int V = 1; //stride_w

	int pad_h = 0;
	int pad_w = 0;
	
	int dilation = 1;

	int P = (H + 2*pad_h - (((R-1)*dilation) + 1) )/U + 1; //output height
	int Q = (W + 2*pad_w - (((S-1)*dilation) + 1) )/U + 1; //output width

/*
	if (argc != 12){
		printf("Usage: You need to type 11 arguments: N C H W K R S pad_h U P Q\n");
		exit(EXIT_FAILURE);
	}

	int	N = atoi(argv[1]);
	int C = atoi(argv[2]);
	int H = atoi(argv[3]);
	int W = atoi(argv[4]);
	int K = atoi(argv[5]);
	int R = atoi(argv[6]);
	int S = atoi(argv[7]);
	int pad_h = atoi(argv[8]);
	int pad_w = atoi(argv[8]);
	int U = atoi(argv[9]);
	int V = atoi(argv[9]);
	int P = atoi(argv[10]);
	int Q = atoi(argv[11]);

	if (!(R==1 && pad_h==0 && U==1))
		return 1;
*/
	
	//int dilation = 1;
	//prepare data
	float *h_input = (float*) malloc(N*C*H*W*sizeof(float));
	for (int j=0; j<N*C*H*W; ++j)
		h_input[j] = 1.f;

	float *h_filter = (float*) malloc(K*C*R*S*sizeof(float));
	for (int j=0; j<K*C*R*S; ++j)
		h_filter[j] = 1.f;

	float *h_result_cudnn = (float*) malloc(K*P*Q*N*sizeof(float));
	float *h_result_our = (float*) malloc(K*P*Q*N*sizeof(float));

	//cuDNN
	//prepare data
	float *input; //input data
	float *filter; //filter
	float *result_cudnn; //result

	ErrChk(hipMalloc(&input, N*C*H*W*sizeof(float)));
	ErrChk(hipMalloc(&filter, K*C*R*S*sizeof(float)));
	ErrChk(hipMalloc(&result_cudnn, N*K*P*Q*sizeof(float)));
	
	ErrChk(hipMemcpy(input, h_input, N*C*H*W*sizeof(float), hipMemcpyHostToDevice));
	ErrChk(hipMemcpy(filter, h_filter, K*C*R*S*sizeof(float), hipMemcpyHostToDevice));
	
	float one = 1.0, zero = 0.0;
	size_t size;

	hipdnnHandle_t handle;
	ErrChk(hipdnnCreate(&handle));


	hipdnnTensorDescriptor_t xDesc, yDesc;
	hipdnnFilterDescriptor_t filterDesc; // HIPDNN_TENSOR_NHWC, HIPDNN_TENSOR_NCHW
	ErrChk(hipdnnCreateTensorDescriptor(&xDesc));
	ErrChk(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

	ErrChk(hipdnnCreateTensorDescriptor(&yDesc));
	ErrChk(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, K, P, Q));

	ErrChk(hipdnnCreateFilterDescriptor(&filterDesc));
	ErrChk(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, K, C, R, S));

	hipdnnConvolutionDescriptor_t convDesc;
	ErrChk(hipdnnCreateConvolutionDescriptor(&convDesc));
	ErrChk(hipdnnSetConvolution2dDescriptor(convDesc, pad_h, pad_w, U, V, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

	hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;

	ErrChk(hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, filterDesc, convDesc, yDesc, algo, (size_t *)&(size)));

	float *extra;
	ErrChk(hipMalloc((void **) &extra, size));

	//  3. Computing
	ErrChk(hipdnnConvolutionForward(handle, &one, xDesc, input, filterDesc, filter, convDesc, algo, extra, size, &zero, yDesc, result_cudnn));
	
	ErrChk(hipMemcpy(h_result_cudnn, result_cudnn, sizeof(float)*N*K*P*Q, hipMemcpyDeviceToHost));



	//Our implementation
	//matrix parameters, because the matrix is stored in Row-Major style and MM is Column-Major, A*B -> BT * AT
		
	float *result_our;
	ErrChk(hipMalloc((void**)&result_our, N*K*P*Q*sizeof(float)));
	

	//gemm	1101
	dim3 block_size;
	block_size.x = 64;
	block_size.y = 1;
	block_size.z = 1;
	
	dim3 grid_size;
	grid_size.x = K/16;
	grid_size.y = (Q*P-1)/16 + 1;
	grid_size.z = N;

	if (H*W%2)
		implicit_gemm_1101_1<<<grid_size, block_size>>>(input, filter, result_our, N, C, H, W, K);
	else if (H*W%16)
		implicit_gemm_1101_2<<<grid_size, block_size>>>(input, filter, result_our, N, C, H, W, K);
	else
		implicit_gemm_1101_16<<<grid_size, block_size>>>(input, filter, result_our, N, C, H, W, K);
	
 	KernelErrChk();

	ErrChk(hipMemcpy(h_result_our, result_our, sizeof(float)*N*K*P*Q, hipMemcpyDeviceToHost));
	
	//Result Test
	for (int j=0; j<N*K; ++j){
		for (int i=0; i<P*Q; ++i)
			printf("%.f ", h_result_cudnn[j*P*Q+i]);
		printf("\n");
	}
	printf("\n");

	printf("----------------------------------\n");
	for (int j=0; j<N*K; ++j){
		for (int i=0; i<P*Q; ++i)
			printf("%.f ", h_result_our[j*P*Q+i]);
		printf("\n");
	}
	printf("\n");

	for (int j=0; j<N*K*P*Q; ++j){
		if (abs(h_result_cudnn[j] - h_result_our[j]) > 10e-2){
			printf("Rejected @ %d\n", j);
			exit(EXIT_FAILURE);
		}
	}
	printf("Passed\n");

	
	ErrChk(hipdnnDestroy(handle));
	ErrChk(hipdnnDestroyTensorDescriptor(xDesc));
	ErrChk(hipdnnDestroyTensorDescriptor(yDesc));
	ErrChk(hipdnnDestroyFilterDescriptor(filterDesc));
	ErrChk(hipdnnDestroyConvolutionDescriptor(convDesc));

	ErrChk(hipFree(input));
	ErrChk(hipFree(filter));
	ErrChk(hipFree(result_our));
	ErrChk(hipFree(result_cudnn));
	ErrChk(hipFree(extra));
	
	free(h_input);
	free(h_filter);
	free(h_result_our);
	free(h_result_cudnn);


	return 0;
}
